#include <hipfft/hipfft.h>       
#include <stdio.h>

#include "cufft_benchmark.h"

#define NX (1LL<<23)
#define BATCH 10

hipfftComplex *data;

int main()
{
    CHECK(hipMalloc((void**)&data, sizeof(hipfftComplex)*NX*BATCH));
    printf("size: %d\n", sizeof(hipfftComplex)*NX*BATCH);

    hipfftHandle plan;
    printf("cufft plan result: %d\n",hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH));

    start_time();

    for(int i=0;i<100;i++)
        printf("cufft execute result: %d\n",hipfftExecC2C(plan, data, data, HIPFFT_FORWARD));
    hipDeviceSynchronize();
    end_time("float32 cufft time cost: ");

    hipfftDestroy(plan);
    hipFree(data);

    return 0;
}

