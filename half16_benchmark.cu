#include <stdio.h>

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>

#include "cufft_benchmark.h"

#define NX (1LL<<23)
#define BATCH 10

half2 *data; 

int main()
{
    CHECK(hipMalloc((void**)&data, sizeof(half2)*NX*BATCH));
    printf("size: %d\n", sizeof(half2)*NX*BATCH);

    long long sample_size = NX;
    int block_size = BATCH;
    size_t workSize = 0;

    hipfftHandle plan_inverse;
    hipfftCreate(&plan_inverse);
    printf("cufftXt plan result: %d\n", hipfftXtMakePlanMany(plan_inverse,
        1, &sample_size,
        NULL, 1, 1, HIP_C_16F,
        NULL, 1, 1, HIP_C_16F,
        block_size, &workSize, HIP_C_16F));

    start_time();

    for(int i=0;i<100;i++)
        printf("cufftXt execute result: %d\n", hipfftXtExec(plan_inverse, data, data, HIPFFT_FORWARD));
    hipDeviceSynchronize();
    end_time("half16 cufft time cost: ");

    hipfftDestroy(plan_inverse);
    hipFree(data);

    return 0;
}

